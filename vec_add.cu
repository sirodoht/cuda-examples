// vec_add.cu: Parallel vector add using CUDA

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


// Kernel function, runs on GPU
__global__ void add_vectors(float *a, float *b, float *c) {
		int i = blockIdx.x;
		
		c[i] = a[i] + b[i];
}


int main(void) {
		int count, i;
		
		// Find number of GPUs
		hipGetDeviceCount(&count);
		printf("There are %d GPU devices in your system\n", count);

		int N = 10;	// Vector length

    // Create vectors a, b and c in the host (CPU)
		float *a = (float *)malloc(N*sizeof(float));
		float *b = (float *)malloc(N*sizeof(float));
		float *c = (float *)malloc(N*sizeof(float));
    
		// Initialize a and b
		for (i=0; i<N; i++) {
			a[i] = i - 0.5;
			b[i] = i*i - 3;
		}

		// Create a_dev, b_dev, c_dev on GPU
		float *a_dev, *b_dev, *c_dev;
		hipMalloc((void **)&a_dev, N*sizeof(float));
		hipMalloc((void **)&b_dev, N*sizeof(float));
		hipMalloc((void **)&c_dev, N*sizeof(float));
		
    // Copy a, b and c vectors from host to GPU
		hipMemcpy(a_dev, a, N*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(b_dev, b, N*sizeof(float), hipMemcpyHostToDevice);
		
    // Parallel add c_dev[i] = a_dev[i] + b_dev[i]
		add_vectors<<< N, 1 >>>(a_dev, b_dev, c_dev);

		// Copy result from GPU to host (CPU)
		hipMemcpy(c, c_dev, N*sizeof(float), hipMemcpyDeviceToHost);
		
    // Free memory
		hipFree(a_dev);
		hipFree(b_dev);
		hipFree(c_dev);

		// Print result on host (CPU)
		printf("\nVector Addition Result:\n");
		for (i=0; i<N; i++) {
			printf("a[%d] : %0.2f \t+\t", i, a[i]);
			printf("b[%d] : %0.2f \t=\t", i, b[i]);
			printf("c[%d] : %0.2f\n", i, c[i]);
		}
		
		return 0;
}
